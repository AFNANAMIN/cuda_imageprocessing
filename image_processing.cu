#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
using namespace std;

#define  i_size 6//image size
#define o_size 6
#define k_size 3//kernel size
int input[i_size][i_size];
int kernel[k_size][k_size];
int output[i_size][i_size];
typedef int itype[i_size];
typedef int ktype[k_size];
void fill_image(int m[i_size][i_size]) {
	static int n = 0; int i, j;
	for (i = 0; i < i_size; i++)
		for (j = 0; j < i_size; j++)
			m[i][j] = n++;
}

void fill_kernel(int m[k_size][k_size]) {
	static int n = 0; int i, j;
	for (i = 0; i < k_size; i++)
		for (j = 0; j < k_size; j++)
			m[i][j] = n++;
}
void fill_output(int m[i_size][i_size]) {
	int i, j;

	for (i = 0; i < o_size; i++) {
		cout << "\n \t\t |";
		for (j = 0; j < o_size; j++)
			cout << "\t\t" << m[i][j];
		cout << "|";
	}

}
__global__ void add_arrays_gpu(int* a, int *b, int* c)
{
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}
__global__ void processing(itype  *a,ktype *kernel, itype *o)
{
	
	int r = 0;
	int i = (blockIdx.y*blockDim.y + threadIdx.y)+1;
	int j = (blockIdx.x*blockDim.x + threadIdx.x)+1;
	
	
	for (int k = -1; k < 2; k++)
	{
		for (int m = -1; m < 2; m++)
		{

			r += a[i + k][j + m] * kernel[k + 1][m + 1];
			o[i][j] = r;
		}

	}
}




int main()
{fill_image(input);
fill_kernel(kernel);
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);


	itype *device_a, *device_c;
	ktype *device_b;
	const int size = i_size*i_size * sizeof(int);
	
	size_t pitch;
	hipMallocPitch((void**)&device_a, &pitch, i_size * sizeof(float), i_size);
	hipMallocPitch((void**)&device_b, &pitch, k_size * sizeof(float), k_size);
	hipMallocPitch((void**)&device_c, &pitch, i_size * sizeof(float), i_size);
	dim3 blockspergrid(2,2,1);
	dim3 threadperblock(2,2,1);
	hipMemcpy(
		device_a, input,
		size,
		hipMemcpyHostToDevice
	);

	hipMemcpy(
		device_b, kernel,
		size,
		hipMemcpyHostToDevice
	);
	int r = 0;
	//add_arra<< <1, count >> > (device_a, device_b, device_c);
	hipEventRecord(start);
	//processing <<<blockspergrid, threadperblock >> > (device_a,device_b,device_c);
	for (int i = 1; i < i_size - 1; i++)
	{
		for (int j = 1; j < i_size - 1; j++)
			for (int k = -1; k < 2; k++)
			{
				for (int m = -1; m < 2; m++)
				{

					r += input[i + k][j + m] * kernel[k + 1][m + 1];
					output[i][j] = r;
				}

			}
	}




	hipEventRecord(stop);
	
	/*cudaMemcpy(
		output, device_c,
		size,
		cudaMemcpyDeviceToHost
	);*/
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "\n total time to execute operation is " << milliseconds << "\n";
	fill_output(output);
	

	//getchar();

	return 0;
}